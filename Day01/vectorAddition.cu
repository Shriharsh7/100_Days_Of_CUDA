#include "hip/hip_runtime.h"
%%writefile vector_add.cu

#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addVectors(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        c[i] = a[i] + b[i];
    }
}

int main(){
    const int n = 1000000;  // Vector size
    int *ha, *hb, *hc;      // Host pointers (CPU)
    int *da, *db, *dc;      // Device pointers (GPU)

    // Allocate memory on host (CPU)
    ha = new int[n];
    hb = new int[n];
    hc = new int[n];

    // Initialize vectors on host

    for (int i = 0; i < n; i++) {
        ha[i] = i;
        hb[i] = 2 * i;
    }

    // Allocate memory on device (GPU)

    if (hipMalloc(&da, n * sizeof(int)) != hipSuccess ||
        hipMalloc(&db, n * sizeof(int)) != hipSuccess ||
        hipMalloc(&dc, n * sizeof(int)) != hipSuccess) {
        std::cerr << "Error allocating device memory." << std::endl;
        return 1;
    }

    // Copy data from host to device

    hipMemcpy(da, ha, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, hb, n * sizeof(int), hipMemcpyHostToDevice);

    // Set up grid and block sizes

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel

    addVectors<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, n);

    // Check for errors in kernel launch and execution

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    // Copy the result back from device to host

    hipMemcpy(hc, dc, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print some results to verify
    
    for (int i = 0; i < 10; i++) {
        std::cout << hc[i] << " ";
    }
    std::cout << std::endl;

    // Free memory

    delete[] ha;
    delete[] hb;
    delete[] hc;
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return 0;
}
/*

!nvcc -gencode=arch=compute_75,code=sm_75 -allow-unsupported-compiler vector_add.cu -o vector_add
!./vector_add

*/
