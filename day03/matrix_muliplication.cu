#include "hip/hip_runtime.h"
%%writefile matrix_mutiplication.cu

#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for matrix multiplication
__global__ void matrixMulKernel(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    const int N = 1024;          // Matrix dimension
    const int BLOCK_SIZE = 16;   // Threads per block dimension
    
    float *h_A, *h_B, *h_C; // Host matrices
    float *d_A, *d_B, *d_C; // Device matrices
    size_t size = N * N * sizeof(float);
    
    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    
    // Initialize host matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }
    
    // Allocate device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    // Configure grid/block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1)/BLOCK_SIZE, 
                (N + BLOCK_SIZE - 1)/BLOCK_SIZE);
    
    // Launch kernel
    matrixMulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
    
    // Copy result back
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
    // Verify result
    bool correct = true;
    for (int i = 0; i < N * N; i++) {
        if (fabs(h_C[i] - 2*N) > 1e-5) {
            correct = false;
            break;
        }
    }
    printf(correct ? "Success!\n" : "Failure!\n");
    
    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return 0;
}

/* 
!nvcc -gencode=arch=compute_75,code=sm_75 -allow-unsupported-compiler matrix_mutiplication.cu -o matrix_mutiplication
!./matrix_mutiplication

Success!

*/
